#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLK 32
#define dimA (10*BLK*10*BLK)
#define dimB (10*BLK*20*BLK)
#define szA (10*BLK*10*BLK*sizeof(double))
#define szB (10*BLK*20*BLK*sizeof(double))
#define A ((const double (*)[10*BLK])a)
#define B ((const double (*)[20*BLK])b)
#define C ((double (*)[20*BLK])c)
#define bx blockIdx.x
#define by blockIdx.y
#define tx threadIdx.x
#define ty threadIdx.y

void init(int n,double *M){
    int i;
    for(i=0;i<n;i++){
        M[i]=(double)rand()/RAND_MAX;
    }
}

void host_mm(const double *a,const double *b,double *c){
    int i,j,k;
    for(i=0;i<10*BLK;i++){
        for(j=0;j<20*BLK;j++){
            for(k=0;k<10*BLK;k++){
                C[i][j]+=A[i][k]*B[k][j];
            }
        }
    }
}

void print(double *c){
    int i,j;
    for(i=0;i<10*BLK;i++){
        for(j=0;j<20*BLK;j++){
            printf("%.2f\t",C[i][j]);
        }
        printf("\n");
    }
}

__global__
void device_mm(const double *a,const double *b,double *c){
    int k;
    for(k=0;k<10*BLK;k++)
        C[bx*BLK+tx][by*BLK+ty]+=A[bx*BLK+tx][k]*B[k][by*BLK+ty];
}

__global__
void tiled_device_mm(const double *a,const double *b,double *c){
    __shared__ double sA[BLK][BLK];
    __shared__ double sB[BLK][BLK];
    int si,i;
    double sum=0;
    for(si=0;si<10;si++){
        sA[tx][ty]=A[bx*BLK+tx][si*BLK+ty];
        sB[tx][ty]=B[si*BLK+tx][by*BLK+ty];
        __syncthreads();
        for(i=0;i<BLK;i++){
            sum+=sA[tx][i]*sB[i][ty];
        }
        __syncthreads();
    }
    C[bx*BLK+tx][by*BLK+ty]=sum;
}

void check(int n,double *x,double *y){
    int i;
    double maxerr=0;
    for(i=0;i<n;i++){
        if(fabsf(x[i]-y[i])/y[i]>maxerr){
            maxerr=fabsf(x[i]-y[i])/y[i];
        }
    }
    printf("max err = %g\n",maxerr);
}

int main(){
    clock_t start,finish;
    double hosttime,devicetime;

    dim3 th(BLK,BLK);
    dim3 bl(10,20);

    double *hA,*hB,*rC,*dA,*dB,*dC,*hC;
    hA=(double*)malloc(szA);
    hB=(double*)malloc(szB);
    hC=(double*)malloc(szB);
    rC=(double*)malloc(szB);

    init(dimA,hA);
    init(dimB,hB);
    memset(hC,0,szB);

    start=clock();
    host_mm(hA,hB,hC);
    finish=clock();
    hosttime=(double)(finish-start)/CLOCKS_PER_SEC;
    printf("host: %.3f\n",hosttime);

    hipMalloc(&dA,szA);
    hipMalloc(&dB,szB);
    hipMalloc(&dC,szB);
    hipMemcpy(dA,hA,szA,hipMemcpyHostToDevice);
    hipMemcpy(dB,hB,szB,hipMemcpyHostToDevice);
    hipMemset(dC,0,szB);
    start=clock();
    device_mm<<<bl,th>>>(dA,dB,dC);
    hipDeviceSynchronize();
    finish=clock();
    hipMemcpy(rC,dC,szB,hipMemcpyDeviceToHost);
    devicetime=(double)(finish-start)/CLOCKS_PER_SEC;
    printf("device: %.3f, speedup=%.3f\n",devicetime,hosttime/devicetime);
    check(dimB,rC,hC);
    //print(rC);
    //printf("CPU:\n");
    //print(hC);

    hipMemcpy(dA,hA,szA,hipMemcpyHostToDevice);
    hipMemcpy(dB,hB,szB,hipMemcpyHostToDevice);
    hipMemset(dC,0,szB);
    start=clock();
    tiled_device_mm<<<bl,th>>>(dA,dB,dC);
    hipDeviceSynchronize();
    finish=clock();
    hipMemcpy(rC,dC,szB,hipMemcpyDeviceToHost);
    devicetime=(double)(finish-start)/CLOCKS_PER_SEC;
    printf("tiled_device: %.3f, speedup=%.3f\n",devicetime,hosttime/devicetime);
    check(dimB,rC,hC);

    free(hA);
    free(hB);
    free(hC);
    free(rC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}